// main.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_cuda<<<2, 2>>>();
    hipDeviceSynchronize();
    std::cout << "Hello from CPU!" << std::endl;
    return 0;
}
